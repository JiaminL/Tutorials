
/* asum: sum of all entries of a vector.
 * This code only calculates one block to show the usage of shared memory and synchronization */

#include <stdio.h>
#include <hip/hip_runtime.h>

typedef double FLOAT;

/* sum all entries in x and asign to y */
__global__ void VecSumKnl(const FLOAT *x, FLOAT *y)
{
    __shared__ FLOAT sdata[256];
    int tid = threadIdx.x;

    /* load data to shared mem */
    sdata[tid] = x[tid];
    __syncthreads();

    /* reduction using shared mem */
    if (tid < 128) sdata[tid] += sdata[tid + 128];
    __syncthreads();

    if (tid < 64) sdata[tid] += sdata[tid + 64];
    __syncthreads();

    if (tid < 32) sdata[tid] += sdata[tid + 32];
    __syncthreads();

    if (tid < 16) sdata[tid] += sdata[tid + 16];
    __syncthreads();

    if (tid < 8) sdata[tid] += sdata[tid + 8];
    __syncthreads();

    if (tid < 4) sdata[tid] += sdata[tid + 4];
    __syncthreads();

    if (tid < 2) sdata[tid] += sdata[tid + 2];
    __syncthreads();

    if (tid == 0) {
        *y = sdata[0] + sdata[1];
    }
}

int main()
{
    int N = 256;   /* must be 256 */
    int nbytes = N * sizeof(FLOAT);

    FLOAT *dx = NULL, *hx = NULL;
    FLOAT *dy = NULL;
    int i;
    FLOAT as = 0;

    /* allocate GPU mem */
    hipMalloc((void **)&dx, nbytes);
    hipMalloc((void **)&dy, sizeof(FLOAT));

    if (dx == NULL || dy == NULL) {
        printf("couldn't allocate GPU memory\n");
        return -1;
    }

    printf("allocated %e MB on GPU\n", nbytes / (1024.f * 1024.f));

    /* alllocate CPU mem */
    hx = (FLOAT *) malloc(nbytes);

    if (hx == NULL) {
        printf("couldn't allocate CPU memory\n");
        return -2;
    }
    printf("allocated %e MB on CPU\n", nbytes / (1024.f * 1024.f));

    /* init */
    for (i = 0; i < N; i++) {
        hx[i] = 1;
    }

    /* copy data to GPU */
    hipMemcpy(dx, hx, nbytes, hipMemcpyHostToDevice);

    /* call GPU */
    VecSumKnl<<<1, N>>>(dx, dy);

    /* let GPU finish */
    hipDeviceSynchronize();

    /* copy data from GPU */
    hipMemcpy(&as, dy, sizeof(FLOAT), hipMemcpyDeviceToHost);

    printf("VecSumKnl, answer: 256, calculated by GPU:%g\n", as);

    hipFree(dx);
    hipFree(dy);
    free(hx);

    return 0;
}
